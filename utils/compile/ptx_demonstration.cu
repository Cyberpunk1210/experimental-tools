#include <iostream>
#include <hip/hip_runtime.h>


__global__ void kernel(int x) {
    asm(".reg .u32 t1;\n\t"
        " mul.lo.u32 t1, %1, %1;\n\t"
        " mul.lo.u32 %0, t1, %1;"
        : "=r"(x) : "r"(x));
    printf("x=%d\n", x);
}

int main()
{
    unsigned int x = 5;
    kernel<<<1, 1>>>(x);
    hipDeviceSynchronize();

    std::cout << "PTX code embedded successfully!" << std::endl;
    return 0;
}